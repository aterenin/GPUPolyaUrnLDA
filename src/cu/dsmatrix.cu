#include "dsmatrix.cuh"
#include "error.cuh"
#include "train.cuh"

namespace gpulda {

template<class T>
DSMatrix<T>::DSMatrix<T>() {
  hipMalloc/*Pitch*/(&dense, args->K * args->V * sizeof(T)) >> GPULDA_CHECK;
}

template<class T>
DSMatrix<T>::~DSMatrix<T>() {
  hipFree(dense) >> GPULDA_CHECK;
}

template class DSMatrix<f32>;
template class DSMatrix<u32>;

}
