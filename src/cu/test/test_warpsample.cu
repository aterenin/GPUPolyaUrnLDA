#include "test_warpsample.cuh"
#include "../warpsample.cuh"
#include "../error.cuh"
#include "assert.h"

using gpulda::FileLine;
using gpulda::f32;
using gpulda::i32;
using gpulda::u32;
using gpulda::u64;

namespace gpulda_test {

void test_compute_d_idx() {
  u32 size = 4*GPULDA_COMPUTE_D_IDX_BLOCKDIM;
  u32 d_len[4*GPULDA_COMPUTE_D_IDX_BLOCKDIM];
  u32 d_idx[4*GPULDA_COMPUTE_D_IDX_BLOCKDIM];
  u32 n_docs = 2*GPULDA_COMPUTE_D_IDX_BLOCKDIM + 15;

  for(i32 i = 0; i < size; ++i) {
    d_len[i] = i+1;
  }

  u32* gpu_d_len;
  u32* gpu_d_idx;
  hipMalloc(&gpu_d_len, size*sizeof(u32)) >> GPULDA_CHECK;
  hipMalloc(&gpu_d_idx, size*sizeof(u32)) >> GPULDA_CHECK;

  hipMemcpy(gpu_d_len, d_len, size*sizeof(u32), hipMemcpyHostToDevice) >> GPULDA_CHECK;

  gpulda::compute_d_idx<<<1,GPULDA_COMPUTE_D_IDX_BLOCKDIM>>>(gpu_d_len, gpu_d_idx, n_docs);
  hipDeviceSynchronize() >> GPULDA_CHECK;

  hipMemcpy(d_idx, gpu_d_idx, size*sizeof(u32), hipMemcpyDeviceToHost) >> GPULDA_CHECK;

  assert(d_idx[0] == 0);
  u32 j = d_len[0];
  for(i32 i = 1; i < n_docs; ++i) {
     assert(d_idx[i] == j);
     j = j + d_len[i];
   }

  hipFree(gpu_d_len);
  hipFree(gpu_d_idx);
}

void test_warp_sample_topics() {

}

}
